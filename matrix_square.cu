
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 1024

void debug_print(int size, int *mat)
{
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("%d ", mat[i * size + j]);
        }

        printf("\n");
    }
}

__global__ void matrix_square(int size, int *res, int *mat)
{
    int i = blockIdx.x;
    int j = threadIdx.x;

    res[i * size + j] = 0;

    for (int k = 0; k < size; k++)
    {
        res[i * size + j] += mat[i * size + k] * mat[k * size + j];
    }
}

int main()
{
    int *hmat, *hres;
    int *mat, *res;

    hmat = (int *)malloc(sizeof(int) * SIZE * SIZE);
    hres = (int *)malloc(sizeof(int) * SIZE * SIZE);

    for (int i = 0; i < SIZE; i++)
    {
        for (int j = 0; j < SIZE; j++)
        {
            hmat[i * SIZE + j] = i * SIZE + j;
        }
    }

    hipMalloc(&mat, sizeof(int) * SIZE * SIZE);
    hipMalloc(&res, sizeof(int) * SIZE * SIZE);

    hipMemcpy(mat, hmat, sizeof(int) * SIZE * SIZE, hipMemcpyHostToDevice);

    matrix_square<<<SIZE, SIZE>>>(SIZE, res, mat);

    hipMemcpy(hres, res, sizeof(int) * SIZE * SIZE, hipMemcpyDeviceToHost);

    debug_print(SIZE, hres);

    hipFree(mat);
    hipFree(res);

    free(hmat);
    free(hres);

    return 0;
}
