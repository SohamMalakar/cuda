
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world()
{
    printf("Hello, world!\n");
}

int main()
{
    hello_world<<<32, 32>>>();
    hipDeviceSynchronize();
    return 0;
}
